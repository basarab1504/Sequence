﻿#include "hip/hip_runtime.h"

#include <ctime>
#include <stdio.h>
#include <iostream>
#include <string>

__global__ void squareKernel(int* data);

int main(int argc, char** argv)
{
	int* h_data;
	int* d_data;
	//количество квадратов + 1
	int n = 10;
	std::string name;

	// выделяем page-locked память на хосте
	// эту функцию лучше всего использовать экономно для выделения промежуточных областей для обмена данными между хостом и устройством.
	hipHostAlloc(&h_data, n * sizeof(int), hipHostMallocPortable);

	//cudaMemcpy(h_data, arr, n * sizeof(int), cudaMemcpyHostToDevice);

	// выделяем память на устройстве
	hipMalloc(&d_data, n * sizeof(int));

	dim3 block(512);
	dim3 grid((n + block.x - 1) / block.x);

	//grid - количество блоков
	//block - размер блока
	squareKernel<<<grid, block>>>(d_data);

	//копируем данные с устройства (d_data) на хост (h_data)
	hipMemcpy(h_data, d_data, n * sizeof(int), hipMemcpyDeviceToHost);

	for (int j = 0; j < n; j++)
	{
		name += std::to_string(h_data[j]);
	}

	for (int j = 1; j < name.size()-1; j+=2)
	{
		std::cout << name[j] << name[j+1] << std::endl;
	}

	return 0;
}

__global__ void squareKernel(int* data)
{
	//threadIdx – номер нити в блоке
	//blockIdx – номер блока, в котором находится нить
	//blockDim – размер блока

	//глобальный индекс нити внутри сети
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	data[i] = powf(2, threadIdx.x);
}
